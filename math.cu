#include "hip/hip_runtime.h"
#include "math.cuh"

CUDA_HOSTDEV Vector::Vector(double xPos, double yPos, double zPos) : p{xPos, yPos, zPos} {}

CUDA_HOSTDEV Vector::Vector(const Point &point) : p{point} {}

double CUDA_HOSTDEV Vector::getX() const {
    return p.x;
}

double CUDA_HOSTDEV Vector::getY() const {
    return p.y;
}

double CUDA_HOSTDEV Vector::getZ() const {
    return p.z;
}

double CUDA_HOSTDEV dot(const Vector& v1, const Vector& v2) {
    return v1.getX() * v2.getX() + v1.getY() * v2.getY() + v1.getZ() * v2.getZ();
}

Vector CUDA_HOSTDEV computeSphereNormal(const Point &p1, const Point &c) {
    Vector normal(p1 - c);
    normal.normalize();
    return normal;
}

double CUDA_HOSTDEV clamp(double value, double min_value, double max_value) {
    return max(min_value, std::min(value, max_value));
}

Point CUDA_HOSTDEV Vector::getVectorPoint() const {return p;}

double CUDA_HOSTDEV Vector::mag() const {
    return sqrt(pow(p.x, 2) + pow(p.y, 2) + pow(p.z, 2));
}
void CUDA_HOSTDEV Vector::normalize() {
    double mag = this->mag();
    if (mag == 0.0) {
        printf("Warning: Cannot normalize a zero vector\n");
    }
    p.x /= mag;
    p.y /= mag;
    p.z /= mag;
}

CUDA_HOSTDEV BVHNode* buildBVH(Sphere* spheres, int start, int end, int depth) {
    BVHNode* node = new BVHNode();

    // Compute bounding box for current node
    for (int i = start; i < end; ++i) {
        node->bbox.expandToInclude(spheres[i]);
    }

    int numSpheres = end - start;
    if (numSpheres == 1) {
        // Leaf node
        node->spheres = &spheres[start];
        node->sphereCount = numSpheres;
    } else {
        int axis = depth % 3;
        std::nth_element(&spheres[start], &spheres[start + numSpheres / 2], 
                         &spheres[end], [axis](const Sphere& a, const Sphere& b) {
                             if (axis == 0)
                                 return a.c.x < b.c.x;
                             else if (axis == 1)
                                 return a.c.y < b.c.y;
                             else
                                 return a.c.z < b.c.z;
                         });

        int mid = start + numSpheres / 2;
        node->left = buildBVH(spheres, start, mid, depth + 1);
        node->right = buildBVH(spheres, mid, end, depth + 1);
    }

    return node;
}

bool CUDA_HOSTDEV intersectSphere(Ray& ray, Sphere& sphere, Intersection& intersection) {
    Vector diff(sphere.c - ray.origin);
    bool inside = pow(diff.mag(), 2.0) < pow(sphere.r, 2.0);
    double tc = dot(diff, ray.direction) / ray.direction.mag();
    if (!inside && tc < 0) false;
    Point d = ray.origin + tc * ray.direction.getVectorPoint() - sphere.c;
    double d2 = pow(Vector(d).mag(), 2.0);
    if (!inside && pow(sphere.r, 2.0) < d2) return false;
    double tOffset = sqrt(pow(sphere.r, 2) - d2) / ray.direction.mag();
    double t = 0.0;
    intersection.found = true;
    t = inside ? tc + tOffset : tc - tOffset;
    if (t < intersection.t) {
        intersection.t = t;
        intersection.c = sphere.color;
        intersection.center = sphere.c;
    }
    if (intersection.found == true) 
        intersection.p = intersection.t * ray.direction.getVectorPoint() + ray.origin;
    return intersection.found == true && intersection.t > 0.0;
}

CUDA_HOSTDEV bool intersectBVH(const BVHNode* node, Ray& ray, Intersection& closestIntersection) {
    if (node == nullptr) return false;

    double tMin = 0.0, tMax = DBL_MAX;
    if (!node->bbox.intersect(ray, tMin, tMax)) {
        return false; 
    }

    if (node->isLeaf()) {
        bool hit = false;
        for (int i = 0; i < node->sphereCount; ++i) {
            if (intersectSphere(ray, node->spheres[i], closestIntersection)) {
                hit = true;
            }
        }
        return hit;
    } else {
        bool hitLeft = intersectBVH(node->left, ray, closestIntersection);
        bool hitRight = intersectBVH(node->right, ray, closestIntersection);
        return hitLeft || hitRight;
    }
}

