#include "hip/hip_runtime.h"
#include "math.cuh"

CUDA_HOSTDEV Vector::Vector(double xPos, double yPos, double zPos) : p{xPos, yPos, zPos} {}

CUDA_HOSTDEV Vector::Vector(const Point &point) : p{point} {}

double CUDA_HOSTDEV Vector::getX() const {
    return p.x;
}

double CUDA_HOSTDEV Vector::getY() const {
    return p.y;
}

double CUDA_HOSTDEV Vector::getZ() const {
    return p.z;
}

double CUDA_HOSTDEV dot(const Vector& v1, const Vector& v2) {
    return v1.getX() * v2.getX() + v1.getY() * v2.getY() + v1.getZ() * v2.getZ();
}

Vector CUDA_HOSTDEV computeSphereNormal(const Point &p1, const Point &c) {
    Vector normal(p1 - c);
    normal.normalize();
    return normal;
}

double CUDA_HOSTDEV clamp(double value, double min_value, double max_value) {
    return max(min_value, std::min(value, max_value));
}

Point CUDA_HOSTDEV Vector::getVectorPoint() const {return p;}

double CUDA_HOSTDEV Vector::mag() const {
    return sqrt(pow(p.x, 2) + pow(p.y, 2) + pow(p.z, 2));
}
void CUDA_HOSTDEV Vector::normalize() {
    double mag = this->mag();
    if (mag == 0.0) {
        printf("Warning: Cannot normalize a zero vector\n");
    }

    p.x /= mag;
    p.y /= mag;
    p.z /= mag;
}

CUDA_HOSTDEV BVHNode* buildBVH(Sphere* spheres, int start, int end, int depth) {
    BVHNode* node = new BVHNode();

    // Compute bounding box for current node
    for (int i = start; i < end; ++i) {
        node->bbox.expandToInclude(spheres[i]);
    }

    int numSpheres = end - start;
    if (numSpheres == 1) {
        // Leaf node
        node->spheres = &spheres[start];
        node->sphereCount = numSpheres;
    } else {
        int axis = depth % 3;
        std::nth_element(&spheres[start], &spheres[start + numSpheres / 2], 
                         &spheres[end], [axis](const Sphere& a, const Sphere& b) {
                             if (axis == 0)
                                 return a.c.x < b.c.x;
                             else if (axis == 1)
                                 return a.c.y < b.c.y;
                             else
                                 return a.c.z < b.c.z;
                         });

        int mid = start + numSpheres / 2;
        node->left = buildBVH(spheres, start, mid, depth + 1);
        node->right = buildBVH(spheres, mid, end, depth + 1);
    }

    return node;
}
